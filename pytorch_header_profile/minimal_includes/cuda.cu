#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <c10/core/TensorOptions.h>
#include <ATen/core/TensorBody.h>
#include <c10/util/ArrayRef.h>
torch::Tensor to_gray(torch::Tensor input) {
  auto output = torch::empty({input.size(0), input.size(1)}, input.options());  
  return output; 
}
