#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

torch::Tensor to_gray(torch::Tensor input) {
  auto output = torch::empty({input.size(0), input.size(1)}, input.options());  
  return output; 
}
